#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdlib.h>
#include <fstream>
#include <string>
#include <string.h>
#include <iostream>
#include <math.h>
#include <assert.h>
#include <fstream>
#include "hip/hip_runtime.h"
#include ""
//#include "hip/hip_runtime_api.h"
#include "hip/hip_complex.h"
//#include <math_functions.h>
#include <hipfft/hipfft.h>
#include "gputimer.h"
#include "gpuerrors.h"
#include "reconst.h"

#define PI 3.141592653589793115997963468544185161590576171875f

// ===========================> Functions Prototype <===============================
void fill(hipFloatComplex* data, int size);
void gpuKernel(hipFloatComplex* in, hipFloatComplex* out);
void printmat(hipFloatComplex* data,int row,int col, char* title);
int read_data(hipFloatComplex* data,char* address);
void write_data(hipFloatComplex* data,char* address, int length);

hipfftHandle handle;
int rank = 1;                           // --- 1D FFTs
int n[] = { Nz_padded };                 // --- Size of the Fourier transform
int istride = 1, ostride = 1;           // --- Distance between two successive input/output elements
int idist = Nz_padded, odist = (Nz_padded); // --- Distance between batches
int inembed[] = { 0 };                  // --- Input size with pitch (ignored for 1D transforms)
int onembed[] = { 0 };                  // --- Output size with pitch (ignored for 1D transforms)
int batch = Nf;                      // --- Number of batched executions

int main(int argc, char** argv) {
	struct hipDeviceProp_t p;
	hipGetDeviceProperties(&p, 0);
	printf("Device Name: %s\n", p.name);
	
	hipFloatComplex* data;
	data = (hipFloatComplex*)malloc(Nf*Nz_padded*Nphi * sizeof(hipFloatComplex));
	//fill(data, Nf*Nz*Nphi);
	
	int readDT = read_data(data, "./data.dat");
	printf("No. of Data Read: %d\n", readDT);
	
	hipFloatComplex* out;
	out = (hipFloatComplex*)malloc(Nz_padded*Ny*Nx * sizeof(hipFloatComplex));

	hipfftPlanMany(&handle, rank, n, 
		inembed, istride, idist,
		onembed, ostride, odist, HIPFFT_C2C, batch);
	
	//time measurement for GPU calculations
	clock_t t1 = clock();
	gpuKernel (data, out);
	clock_t t2 = clock();
	
	//printmat(temph,row+row/2,n,"temph");
	//printmat(out,m,n,"output Matrix");
	
	printf("Execution Time: %ld\n", (t2-t1)/1000);
	
	write_data(out, "./out.dat", Nz_padded*Ny*Nx);
	
	free(out);
	free(data);
	return 0;
}

int read_data(hipFloatComplex* data,char* address)
{
	unsigned int num=0;
	std::ifstream infile(address);
	std::string line;
	char cstr[100];
	char* token;
	unsigned int numline = 0;
	while (std::getline(infile, line))
	{
		if(line.size()>1)
		{
			strcpy(cstr, line.c_str());
			num = 0;
			token = strtok(cstr, ",");
			while (token) {
				assert(num < 2);
				if (num == 0)
				{
					data[numline].x = atof(token);
				} else
				{
					data[numline].y = atof(token);
				}
				token = strtok(NULL, ",");
				num++;
			}
			++numline;
		}
	}
	return numline;
	//printf("Hello, %d!\n",numline);
}

void write_data(hipFloatComplex* data,char* address, int length)
{
	FILE * fp;
	fp = fopen(address, "w");
	if (fp == NULL)
	exit(EXIT_FAILURE);
	for(int i=0; i<length; i++)
	{
		fprintf(fp, "%.6E,%.6E\n", data[i].x, data[i].y);
	}
	fclose(fp);
}

void fill(hipFloatComplex* data, int size) {
	for (int i=0; i<size; ++i)
	{
		data[i].x = (float) (rand() % 10- 5);
		data[i].y = (float) (rand() % 10- 5);
	}
}

void printmat(hipFloatComplex* data,int row,int col, char* title) {
	printf("%s\n",title);
	for (int i=0; i<row; ++i)
	{
		for (int j=0; j<col; ++j)
		{
			printf("%g+%gi\t",mem2d(data,col,i,j).x,mem2d(data,col,i,j).y);
		}
		printf("\n");
	}
	printf("......................\n");
}

void gpuKernel(hipFloatComplex* data, hipFloatComplex* out) {
	hipFloatComplex* voxel1d;
	hipFloatComplex* colData;
	hipFloatComplex* colDataFFT;
	hipFloatComplex Ant_position;
	hipFloatComplex* inp;
	//hipFloatComplex* dtpointer;
	//hipFloatComplex inp [Nz_padded*Nf];
	//memset (inp, 0, Nz_padded*Nf*sizeof(hipFloatComplex));
	//for (int i_f=0; i_f<Nf; i_f++)
	//{
	//	for (int i_z_Ant=0; i_z_Ant<Nz_padded; i_z_Ant++)
	//	{
	//		mem2d(inp,Nz_padded,i_f,i_z_Ant) = make_hipComplex((float) (rand() % 10- 5), (float) (rand() % 10- 5));
	//	}
	//}
	
	HANDLE_ERROR(hipMalloc((void**)&voxel1d, Nz_padded*Ny*Nx * sizeof(hipFloatComplex)));
	HANDLE_ERROR(hipMemset(voxel1d, 0, Nz_padded*Ny*Nx*sizeof(hipFloatComplex)));
	HANDLE_ERROR(hipMalloc((void**)&colData, Nf*Nz_padded* sizeof(hipFloatComplex)));
	HANDLE_ERROR(hipMalloc((void**)&colDataFFT, Nf*Nz_padded* sizeof(hipFloatComplex)));
	
	dim3 dimBlock(Nx);
	dim3 dimGrid(Ny, Nz_padded);
	
	GpuTimer timer;
	timer.Start();
	//printf("%zd\n",sizeof(hipFloatComplex));
	for (int i_phi_Ant=0; i_phi_Ant<Nphi; i_phi_Ant++)
	{
		//printf("if:%d\n",i_phi_Ant);
		Ant_position = make_hipFloatComplex(R_ant*cos( i_phi_Ant * PI / 180.0 )+(Lx/2), R_ant*sin( i_phi_Ant * PI / 180.0 )+(Ly/2));
		//Ant_position = make_hipFloatComplex(R_ant*0.7, R_ant*0.7);
		inp = data + i_phi_Ant*Nz_padded*Nf;
		//HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(colData), inp, sizeof(hipFloatComplex)*Nz*Nf));
		HANDLE_ERROR(hipMemcpy(colData, inp, Nz_padded*Nf*sizeof(hipFloatComplex), hipMemcpyHostToDevice));
		hipfftExecC2C(handle,  colData, colDataFFT, HIPFFT_BACKWARD);
		SARbp<<< dimGrid,dimBlock >>>(voxel1d, colDataFFT, Ant_position);
	}
	timer.Stop();
	float gpu_kernel_time = timer.Elapsed();
	printf("GPU Time:%f\n", gpu_kernel_time);

	HANDLE_ERROR(hipMemcpy(out, voxel1d, Nz_padded*Ny*Nx * sizeof(hipFloatComplex), hipMemcpyDeviceToHost));
	
	HANDLE_ERROR(hipFree(voxel1d));
}

//////////////////////////CUDA\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\\

#define tx threadIdx.x
#define ty threadIdx.y
#define tz threadIdx.z

#define bx blockIdx.x
#define by blockIdx.y
#define bz blockIdx.z

#define X_idx tx
#define Y_idx bx
#define Z_idx by


#define bdx blockDim.x
#define bdy blockDim.y
#define bdz blockDim.z

#define gdx gridDim.x
#define gdy gridDim.y
#define gdz gridDim.z

#define NumThreads 1024
#define DegToRad (PI/180)
#define C 299792458
#define k_prefix (PI*2/C)
#define delta_k (PI*2*delta_f/C)
#define k_0 (k_prefix*(fstart))
#define Nz_padded_inv (1.0f/(Nz_padded-1))
#define k_z_max (PI/dz)
#define init_f_prefix (C/(PI*delta_f*4))
#define f_norm (fstart/delta_f)

#if Prec_SAR==1	
__global__ void SARbp(hipFloatComplex* voxel1d, hipFloatComplex* colData, hipFloatComplex Ant_position)
{	
	const float kz_arg_sqrt = fabsf(k_z_max*(Nz_padded_inv+(2*Nz_padded_inv)*Z_idx-(Z_idx/Nz)*(2*Nz_padded_inv)*(Nz_padded)));
	const float kz_arg = kz_arg_sqrt*kz_arg_sqrt;
	const float distXY = ((Ant_position.x-X_idx*dx)*(Ant_position.x-X_idx*dx) + (Ant_position.y-Y_idx*dy)*(Ant_position.y-Y_idx*dy));
	float exp_arg;
	
	hipFloatComplex voxel, data;
	hipFloatComplex exp_coef;
	
	unsigned long int index = tx + bx*bdx + by*gdx*bdx;
	voxel = make_hipComplex(0,0);

	unsigned int i_f_start = ceilf(init_f_prefix*kz_arg_sqrt - f_norm);
	float f = fstart + i_f_start*delta_f;
	for (int i_f=i_f_start; i_f<Nf; i_f++)
	{
		exp_arg = sqrtf(distXY*(k_prefix*k_prefix*f*f*4 - kz_arg));
		__sincosf(exp_arg, &(exp_coef.y), &(exp_coef.x));
		data = mem2d(colData,Nz_padded,i_f,Z_idx);
		voxel = make_hipComplex(voxel.x + data.x*exp_coef.x - data.y*exp_coef.y, voxel.y + data.x*exp_coef.y + data.y*exp_coef.x);
		f = f+delta_f;
	}
	//voxel1d[index] = voxel;
	atomicAdd(&voxel1d[index].x, voxel.x);
	atomicAdd(&voxel1d[index].y, voxel.y);
}
#else
__global__ void SARbp(hipFloatComplex* voxel1d, hipFloatComplex* colData, hipFloatComplex Ant_position)
{	
	const float kz_arg = (Nz_padded_inv+(2*Nz_padded_inv)*Z_idx-(Z_idx/Nz)*(2*Nz_padded_inv)*(Nz_padded));
	const float dist_kz = sqrt(((Ant_position.x-X_idx*dx)*(Ant_position.x-X_idx*dx) + (Ant_position.y-Y_idx*dy)*(Ant_position.y-Y_idx*dy)) * (4-kz_arg*kz_arg));
	float exp_arg, delta_exp_arg;
	

	hipFloatComplex exp_coef, delta_exp_coef;
	hipFloatComplex voxel, data;
	
	unsigned long int index = tx + bx*bdx + by*gdx*bdx;
	voxel = make_hipComplex(0,0);

	exp_arg = k_0*dist_kz;
	delta_exp_arg = delta_k*dist_kz;
	__sincosf(exp_arg, &(exp_coef.y), &(exp_coef.x));
	__sincosf(delta_exp_arg, &(delta_exp_coef.y), &(delta_exp_coef.x));
	for (int i_f=0; i_f<Nf; i_f++)
	{
		data = mem2d(colData,Nz_padded,i_f,Z_idx);
		voxel = make_hipComplex(voxel.x + data.x*exp_coef.x - data.y*exp_coef.y, voxel.y + data.x*exp_coef.y + data.y*exp_coef.x);
		exp_coef = hipCmulf(exp_coef, delta_exp_coef);
	}
	//voxel1d[index] = voxel;
	atomicAdd(&voxel1d[index].x, voxel.x);
	atomicAdd(&voxel1d[index].y, voxel.y);
}
#endif		
